#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <iostream>
#include <fstream>
#include <sstream> 

// #include <complex>



#include <hipblas.h>  

//#include <cublas.h> 

//-------------------------------------------------------------------------------------

// ---------   Neverno rabotaet "chistoe 3D _cufft" iz-za raznogo razmeshenija dannux
//----------:             SDELANO 2D+ 1D
//       
//--------------------------------------------------------------------------------------

//   cudaMemcpy rabotaet bustree, chem CUBLAS 
//--------------------------------------------------------------------------------------

#define imin(a,b) (a<b?a:b)     


__global__ void  kinetic_3D2pls1( hipDoubleComplex *vec2, 
                             hipDoubleComplex *ax, 
                             hipDoubleComplex *ay, 
                             hipDoubleComplex *az,
                             int NX, int NY, int NZ)
 {    
            
          int kd = threadIdx.x  + blockIdx.x* blockDim.x;

      
      while(kd<NZ)
      {
          for (int id = 0; id < NX; ++id ) 
          {
             for (int jd = 0; jd < NY; ++jd ) 
             {
               
               int ind =id+NX*jd+kd*NX*NY;
              
           hipDoubleComplex sum =hipCadd(ax[id], ay[jd]); 
                           sum= hipCadd( sum, az[kd] );

             vec2[ind]= hipCmul ( sum ,  vec2[ind] );
 
             }
          }
      __syncthreads(); 
        kd += blockDim.x*gridDim.x;

      }
  }
 
  
     void  kinetic_CPU_3D2pls1(   hipDoubleComplex *vec2, 
                             hipDoubleComplex *ax, 
                             hipDoubleComplex *ay, 
                             hipDoubleComplex *az,
                             int NX, int NY, int NZ)
{

         int ind;
         hipDoubleComplex sum;

         for (int idz = 0; idz < NZ; ++idz ) 
         {
            for (int jd = 0; jd < NY; ++jd ) 
            {
              for (int kd = 0;  kd < NX; ++kd ) 
              {
              
             ind = kd+NX*jd+ idz*NX*NY;                   

             sum = hipCadd(ax[kd], ay[jd]); 
             sum = hipCadd( sum,   az[idz] );

             vec2[ind]= hipCmul ( sum,  vec2[ind] );
              }
            }
          }

         return;
}
 

//------------------------------------------------------------------------------------//


//--------------------- function called from main fortran program -------------------//


extern "C" void   kernel_fft_3d2pls1_(hipDoubleComplex *psi,
                                 hipDoubleComplex *ortkx, 
                                 hipDoubleComplex *ortky,
                                 hipDoubleComplex *ortkz, int *NX, int *NY, int *NZ)
     {
       
  //                 printf("Cufft_3D-as 2D plus 1D\n");
//   printf ( " 3d-----------full -----------oksa" ) ;            

          hipDoubleComplex   *ortkx_d, *ortky_d, *ortkz_d;  //, *psiV_d;  // declare GPU vector copies cufftDoubleComplex  *psi_d, *ortkx_d;
    
          hipDoubleComplex   *psi_d;      

           int NDX = *NX;       
           int NDY = *NY;  
           int NDZ = *NZ; 

           int NDIM= NDX*NDY*NDZ; 
           
           hipfftHandle plan2,plan; 

// ---- CUDA  variables: ----------------------------------------------------------------------- 

            const int threads = imin(64,NDIM);   //dim3 (16,1,1);
            const int blocks  = imin(32, NDIM/threads );  // ;  //imin(8, (M+threads)/threads ); 
//----------------------------------------------------------------------------------------------

          hipMalloc( (void **)&psi_d, sizeof(hipDoubleComplex) * NDIM );
    
          hipMalloc( (void **)&ortkx_d, sizeof(hipDoubleComplex) * NDX );
          hipMalloc( (void **)&ortky_d, sizeof(hipDoubleComplex) * NDY );
          hipMalloc( (void **)&ortkz_d, sizeof(hipDoubleComplex) * NDZ );
   


/* --------------- Initialize CUBLAS --------------------------------------- */
// ------------  cublas  -----------------------
// ------------  vozmojno nado allokirovat' memory  cublasAlloc

/*             cublasHandle_t handle;
             
             cublasCreate_v2(&handle);    // cublasCreate_v2(&handle); 

             cublasSetVector(NDIM,sizeof(cuDoubleComplex),psi,1, psi_d,1);          // 1
            
             cublasSetVector(NDX,sizeof(cuDoubleComplex),ortkx,1, ortkx_d,1); 
             cublasSetVector(NDY,sizeof(cuDoubleComplex),ortky,1, ortky_d,1);    
             cublasSetVector(NDZ,sizeof(cuDoubleComplex),ortkz,1, ortkz_d,1); 
*/  
      
 //-----------------------------------------------------------------------------------------------
            
              hipMemcpy( psi_d, psi,     sizeof(hipDoubleComplex) * NDIM, hipMemcpyHostToDevice );
              hipMemcpy( ortkx_d, ortkx, sizeof(hipDoubleComplex) * NDX,  hipMemcpyHostToDevice );
              hipMemcpy( ortky_d, ortky, sizeof(hipDoubleComplex) * NDY,  hipMemcpyHostToDevice ); 
              hipMemcpy( ortkz_d, ortkz, sizeof(hipDoubleComplex) * NDZ,  hipMemcpyHostToDevice ); 
       
  

/* -------------- Create a 2D + 1D FFT plans. ------------------------------------  */
               
                        
                 int num[1], inembed[1], onembed[1];
                   num[0]=NDZ;   
                   inembed[0]= 0; 
                   onembed[0]= 0;  
               
                     int inembed2[2]= {0,0}; 
                     int onembed2[2]= {0,0};      
            
               // cufftPlanMany(&plan,1, num,inembed,1, 0, onembed, 1,0,CUFFT_Z2Z,NDX*NDY); 

                  hipfftPlanMany(&plan,1, num,inembed,NDX*NDY,1, onembed, NDX*NDY,1,HIPFFT_Z2Z,NDX*NDY);
           
              int num2[2]={ NDY, NDX };  //    !!!!!!!!!!!!!!!! pomenjala X<-> Y mestami
               
               hipfftPlanMany(&plan2,2, num2,NULL,1,0,NULL,1,0,HIPFFT_Z2Z, NDZ);
             
                //   cufftPlanMany(&plan2,2, num2,inembed2,0,NDX*NDY, onembed2, 0, NDX*NDY,CUFFT_Z2Z, NDZ);

             //1       cufftPlan2d(&plan2, NDY, NDX, CUFFT_Z2Z); 

/* ------    Use the CUFFT plan to transform the signal in place.  ---------  */
           
         //1 for (int idz = 0; idz < NDZ; ++idz ) 
         //1       {
     
            //1   cufftExecZ2Z(plan2,  &psi_d[idz*NDX*NDY],  &psi_d[idz*NDX*NDY], CUFFT_FORWARD);
              //1  }


                     
                hipfftExecZ2Z(plan2, psi_d, psi_d, HIPFFT_FORWARD);
         
                hipfftExecZ2Z(plan,   psi_d,  psi_d, HIPFFT_FORWARD);

/*---------- call function on GPU -------------------------------------------  */
                 
       
             kinetic_3D2pls1<<<blocks,threads>>>( psi_d, ortkx_d, ortky_d, ortkz_d, NDX, NDY,NDZ);


//----------------------- kernel on CPU -----------------------------------------
//------------------------  PROOF -  -----------------------
   /*    
             cudaMemcpy(psi, psi_d, sizeof(cuDoubleComplex) *NDIM, cudaMemcpyDeviceToHost);
              
              kinetic_CPU_3D2pls1 ( psi, ortkx, ortky, ortkz, NDX, NDY, NDZ);

           
  //         cublasGetVector(NDIM,sizeof(cuDoubleComplex),psi_d,1, psi,1); 
     
            
  
          cudaMemcpy( psi_d, psi,     sizeof(cuDoubleComplex) * NDIM, cudaMemcpyHostToDevice );

//       cublasSetVector(NDIM,sizeof(cuDoubleComplex),psi,1, psi_d,1);
   
  */


//--------------------------------------------------------------------------------------------
   
                hipfftExecZ2Z(plan2, psi_d, psi_d, HIPFFT_BACKWARD);
         
                hipfftExecZ2Z(plan,  psi_d, psi_d, HIPFFT_BACKWARD);

      //1    for (int idz = 0; idz < NDZ; ++idz ) 
         //1       {
     
            //1   cufftExecZ2Z(plan2,  &psi_d[idz*NDX*NDY],  &psi_d[idz*NDX*NDY], CUFFT_INVERSE);
             //1   }



              // printf("Cufft_3D-Inverse\n");

//---------------- copy vector back from GPU to CPU  --------------------------------------------- 
       
         //       cublasGetVector(NDIM,sizeof(cuDoubleComplex),psi_d,1, psi,1);           // 2
               
              hipMemcpy(psi, psi_d, sizeof(hipDoubleComplex) *NDIM, hipMemcpyDeviceToHost); 
            
  
   // free GPU memory
     
           //cublasDestroy_v2(handle);          //  cublasDestroy_v2(handle);
           hipfftDestroy(plan);
           hipfftDestroy(plan2);
  

           hipFree(psi_d);
       
           hipFree(ortkx_d);
           hipFree(ortky_d);
           hipFree(ortkz_d);

                   
        

       return;

         }
