#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>


#include <iostream>

// #include <complex>




#define imin(a,b) (a<b?a:b)     

 

/*--------- function called from main fortran programn ---------------*/
// -------------  Onli 1D, 2D or 3D  cuFFT transform !!!!!!!!!!!!!!!!!!!


extern "C" void kernel_imestfft_t_(hipDoubleComplex *wtilde,int *NX,int *NY,int *NZ,  int *Xcase)
{
           hipDoubleComplex  *psi_d;

        int NDX = *NX;
        int NDY = *NY;
        int NDZ = *NZ;
        int NDIM= NDX*NDY*NDZ;
        int dcase= *Xcase;


//----------------------------------------------------------------------------------------------
//                 printf("IN T GPU _Transform  \n");  

         hipfftHandle plan;
         hipMalloc( (void **)&psi_d, sizeof(hipDoubleComplex) * NDIM );
          hipMemcpy( psi_d, wtilde, sizeof(hipDoubleComplex)*NDIM, hipMemcpyHostToDevice );

 //                printf("Kernel GPU _Transform  \n");  

      if (dcase == 1)  hipfftPlan1d(&plan, NDX, HIPFFT_Z2Z, 1); 
     
     if (dcase == 2)   hipfftPlan2d(&plan, NDY, NDX,      HIPFFT_Z2Z); 
   
     if (dcase == 3)   hipfftPlan3d(&plan, NDZ, NDY, NDX, HIPFFT_Z2Z); 

          hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_FORWARD);

 
        /*  copy vectors from GPU to CPU   */
           
            hipMemcpy(wtilde, psi_d, sizeof(hipDoubleComplex) * NDIM, hipMemcpyDeviceToHost);

           hipfftDestroy(plan);
           hipFree(psi_d);
        return;

}
