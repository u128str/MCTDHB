#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>


#include <iostream>

#include <complex>



#include <hipblas.h>   

 //#include <cublas.h> 
//------------------------------------------------------------------

#define imin(a,b) (a<b?a:b)         


 
 __global__ void kinetic( hipDoubleComplex *vec, hipDoubleComplex *a, int Nx, int Ny, int Nz, int mcase)
{ 
       
      if (mcase==2)
        {
          int idx = threadIdx.x  + blockIdx.x* blockDim.x;

             while(idx<Nx)
             {
              for (int idy = 0; idy < Ny; ++idy ) 
              {
                for (int idz = 0; idz < Nz; ++idz ) 
                {

//            ind=k+(J-1)*NDX+(I-1)*NDX*NDY
//            psi(ind)=psi(ind)*ortky(J)**2/(2*Time_mass)/NDY
               int ind =idx + idy*Nx+idz*Nx*Ny;
              
               vec[ind]=hipCmul(vec[ind],a[idy]);
                 }
              }

            idx += blockDim.x*gridDim.x;
          }
        }

      if (mcase==1)
        {
          int idy = threadIdx.x  + blockIdx.x* blockDim.x;

             while(idy<Ny)
             {
              for (int idz = 0; idz < Nz; ++idz ) 
              {
                for (int idx = 0; idx < Nx; ++idx ) 
                {

               int ind =idx + idy*Nx+idz*Nx*Ny;
              
               vec[ind]=hipCmul(vec[ind],a[idx]);
                 }
              }
//    __syncthreads(); 
            idy += blockDim.x*gridDim.x;
         }
      }
    if (mcase==3)
        {
          int idz = threadIdx.x  + blockIdx.x* blockDim.x;

             while(idz<Nz)
             {
              for (int idx = 0; idx < Nx; ++idx ) 
              {
                for (int idy = 0; idy < Ny; ++idy ) 
                {

               int ind =idx + idy*Nx+idz*Nx*Ny;
              
               vec[ind]=hipCmul(vec[ind],a[idz]);
                 }
              }
//   __syncthreads(); 
            idz += blockDim.x*gridDim.x;
            }
        }
}       

   
  

/*--------- function called from main fortran programn ---------------*/

extern "C" void kernel_fft_1d_(hipDoubleComplex *psi,hipDoubleComplex *ortk,int * Nx,int * Ny,int * Nz, int *Xcase)
{
           hipDoubleComplex  *psi_d, *ortk_d;  // declare GPU vector copies cufftDoubleComplex  *psi_d, *ortkx_d;

     
       int NX = *Nx;
       int NY = *Ny;        
       int NZ = *Nz;
       int NDIM= NX*NY*NZ;
       int ncase= *Xcase;
      
   //    cublasStatus stat ;

//---//----  CUDA variables: 

            const int threads = imin(256,NDIM);   //dim3 (16,1,1);
            const int blocks  = imin(256, NDIM/threads ); 
            hipfftHandle plan;
          
  //        cublasInit ( ) ;
 
       // Allocate memory on GPU
       
          hipMalloc( (void **)&psi_d, sizeof(hipDoubleComplex) * NDIM );

 
  //      cudaMemcpy( psi_d, psi, sizeof(cuDoubleComplex) *NDIM, cudaMemcpyHostToDevice );
        
         //  cublasAlloc (NDIM , sizeof(cuDoubleComplex),(void **)&psi_d ) ;


  /*
         if ( stat != CUBLAS_STATUS_SUCCESS ) 
          {
         printf ( " device memory allocationfailed " ) ;
         cublasShutdown ( ) ;
         return EXIT_FAILURE;
          }
*/

//=====================================================================================          
    if (ncase == 1)
     {
          hipMalloc( (void **)&ortk_d, sizeof(hipDoubleComplex) * NX );
          /* Initialize CUBLAS */
             hipblasHandle_t handle;
             hipblasCreate(&handle);    // cublasCreate_v2(&handle); 
             hipblasSetVector(NDIM,sizeof(hipDoubleComplex),psi,1, psi_d,1);
             hipblasSetVector(NX,sizeof(hipDoubleComplex),ortk,1, ortk_d,1);    
         // cudaMemcpy( ortk_d, ortk, sizeof(cuDoubleComplex) * NX, cudaMemcpyHostToDevice );
         /* Create a 1D FFT plan. */ 
//!cufftPlanMany(cufftHandle *plan, int rank, int *n, int *inembed, int istride, int idist, int *onembed, int ostride, int odist, cufftType type, int batch);
                  int num[1], inembed[1], onembed[1];
                  num[0]=NX;   
                  inembed[0]= 0;  
                  onembed[0]= 0;
              hipfftPlanMany(&plan,1, num,inembed,1,NX, onembed, 1,NX,HIPFFT_Z2Z,NY*NZ);
//               cufftPlanMany(&plan,1, num ,NULL,1,0,NULL,1,0,CUFFT_Z2Z,NY*NZ);
//         printf ( " 1D FFT CUDA CPP in X " ) ;
              hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_FORWARD);
            kinetic<<<blocks,threads>>>(psi_d, ortk_d, NX,NY,NZ, ncase);
               hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_BACKWARD);
        /*  copy vectors from GPU to CPU   */
          hipblasGetVector(NDIM,sizeof(hipDoubleComplex),psi_d,1, psi,1); 
     //         cudaMemcpy(psi, psi_d, sizeof(cuDoubleComplex) * NX, cudaMemcpyDeviceToHost);
           hipblasDestroy(handle);          //  cublasDestroy_v2(handle);
           hipfftDestroy(plan);
           hipFree(psi_d);
           hipFree(ortk_d);
      }
//=====================================================================================          
    if (ncase == 2)
     {
          hipMalloc( (void **)&ortk_d, sizeof(hipDoubleComplex) * NY );
          /* Initialize CUBLAS */
             hipblasHandle_t handle;
             hipblasCreate(&handle);    // cublasCreate_v2(&handle); 
             hipblasSetVector(NDIM,sizeof(hipDoubleComplex),psi,1, psi_d,1);
             hipblasSetVector(NY,sizeof(hipDoubleComplex),ortk,1, ortk_d,1);    
         // cudaMemcpy( ortk_d, ortk, sizeof(cuDoubleComplex) * NX, cudaMemcpyHostToDevice );
         /* Create a 1D FFT plan. */ 


       hipfftResult res;





//!cufftPlanMany(cufftHandle *plan, int rank, int *n, int *inembed, int istride, int idist, int *onembed, int ostride, int odist, cufftType type, int batch);
                 // int num[1]={NY}, inembed[1]={0}, onembed[1]={0};
                     int num[1]={NY}, inembed[1]={0}, onembed[1]={0};

/////             Everything 2D WORKS in 1D along Y direction!!!!
////////            NEpravil'noe rabotaet v 3D po Y direction pochemu??? - hren ego znaet!!!!
//X works              cufftPlanMany(&plan,1, num,inembed,1    ,NX, onembed, 1    ,NX,CUFFT_Z2Z,NY*NZ);
               // res =  cufftPlanMany(&plan,1, num,inembed,NX   ,1 , onembed, NX   ,1 ,CUFFT_Z2Z,NX*NZ);
               
                  res =  hipfftPlanMany(&plan,1, num,inembed,NX,1 , onembed, NX, 1 ,HIPFFT_Z2Z, NX);  // preobrazovanie 1 sloja

//Z works              cufftPlanMany(&plan,1, num,inembed,NX*NY,1 , onembed, NX*NY,1 ,CUFFT_Z2Z,NX*NY);
        //    cufftPlanMany(&plan,1, num,inembed,NX,1, onembed, NX,1,CUFFT_Z2Z,NX*NZ);
//           printf ("1D FFT CUDA CPP in Y res %d \n",res);  
            // cufftPlan1d(&plan, NX, CUFFT_Z2Z, BATCH);
        /* Use the CUFFT plan to transform the signal in place. */

                 for (int idz = 0; idz < NZ; ++idz ) 
                {
     
               hipfftExecZ2Z(plan,  &psi_d[idz*NX*NY],  &psi_d[idz*NX*NY], HIPFFT_FORWARD);
                }

            kinetic<<<blocks,threads>>>(psi_d, ortk_d,  NX,NY,NZ, ncase);
              

                        for (int idz = 0; idz < NZ; ++idz ) 
                {

                 // cufftExecZ2Z(plan,  psi_d,  psi_d, CUFFT_INVERSE);

             hipfftExecZ2Z(plan,  &psi_d[idz*NX*NY],  &psi_d[idz*NX*NY], HIPFFT_BACKWARD);
                }   

 /*          Do J=1,NDY
            Do I=1,NDZ
            Do K=1,NDX
            ind=k+(J-1)*NDX+(I-1)*NDX*NDY
            psi(ind)=psi(ind)*ortky(J)**2/(2*Time_mass)/NDY
            EndDo
            EndDo
            EndDo 
*/
        /*  copy vectors from GPU to CPU   */
          hipblasGetVector(NDIM,sizeof(hipDoubleComplex),psi_d,1, psi,1); 
     //         cudaMemcpy(psi, psi_d, sizeof(cuDoubleComplex) * NX, cudaMemcpyDeviceToHost);
           hipblasDestroy(handle);          //  cublasDestroy_v2(handle);
           hipfftDestroy(plan);
           hipFree(psi_d);
           hipFree(ortk_d);

      }
//======================================================================================== 
   if (ncase == 3)
     {
         hipMalloc( (void **)&ortk_d, sizeof(hipDoubleComplex) * NZ );
            

   // copy vectors from CPU to GPU
      
          /* Initialize CUBLAS */
   
             hipblasHandle_t handle;
             hipblasCreate(&handle);    // cublasCreate_v2(&handle); 
          
             hipblasSetVector(NDIM,sizeof(hipfftDoubleComplex),psi,1, psi_d,1);
             hipblasSetVector(NZ,sizeof(hipfftDoubleComplex),ortk,1, ortk_d,1);    
         
         // cudaMemcpy( ortk_d, ortk, sizeof(cuDoubleComplex) * NX, cudaMemcpyHostToDevice );

         /* Create a 1D FFT plan. */ 

// ETO NE RABOTAET!!!!               cufftPlanMany(&plan,1, Nz,NULL,NX*NY,1,NULL,NX*NY,1,CUFFT_Z2Z,NY*NX);
         //    cufftPlan1d(&plan, NX, CUFFT_Z2Z, BATCH);
                  int num[1], inembed[1], onembed[1];
                  num[0]=NZ;   
                  inembed[0]= 0;  // 0;  oksa: STAVLU NDIM --NE NADo I TAK RABOTAET
                  onembed[0]= 0;      //0;
///   Wrode rabotaet.... v 3D da FFT XZ rabotaet no esli vzyt' FFT Y to vse letot na JUX
              hipfftPlanMany(&plan,1, num,inembed,NX*NY,1, onembed, NX*NY,1,HIPFFT_Z2Z,NX*NY);
//         printf ( " 1D FFT CUDA CPP in Z " ) ;

       
        /* Use the CUFFT plan to transform the signal in place. */
           
               hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_FORWARD);
        
            kinetic<<<blocks,threads>>>(psi_d, ortk_d,  NX,NY,NZ, ncase);
            
               hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_BACKWARD);
        
        /*  copy vectors from GPU to CPU   */
           
          hipblasGetVector(NDIM,sizeof(hipfftDoubleComplex),psi_d,1, psi,1); 
            
     //         cudaMemcpy(psi, psi_d, sizeof(cuDoubleComplex) * NX, cudaMemcpyDeviceToHost);
 
           hipblasDestroy(handle);          //  cublasDestroy_v2(handle);
           hipfftDestroy(plan);

           hipFree(psi_d);
           hipFree(ortk_d);

      }
 


       return;

}
