#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <iostream>
#include <fstream>
#include <sstream> 

// #include <complex>



#include <hipblas.h> 

//<cublas_v2.h>   
                //  old version <cublas.h> 
//-------------------------------------------------------------------------------------

 //---//----  CUDA variables: 

            const int threads = 256;  //dim3 (16,1,1);
            const int blocks  = 16;  //imin(8, (M+threads)/threads ); 


//-----------------------------------------------------------------------------------//

__global__ void mltpl_2D( hipDoubleComplex *vec2, hipDoubleComplex *ax,hipDoubleComplex *ay, int NX, int NY)
   {    
            
          int jd = threadIdx.x  + blockIdx.x* blockDim.x;

          //  for (int jd = 0; jd < NDY; ++jd ) 
    
            while(jd<NY)

            {
             for (int id = 0; id < NX; ++id ) 
             {
               int ind =id+NX*jd;
             vec2[ind]= hipCmul (hipCadd(ax[id], ay[jd] ),  vec2[ind] );
             }
            jd += blockDim.x*gridDim.x;

              __syncthreads(); 

            }
//------------------------------------------------------------------------------------//


   }






// function called from main fortran program

extern "C" void   kernel_fft_2d_(hipfftDoubleComplex *psi,
                hipDoubleComplex *ortkx, hipDoubleComplex  *ortky, int *NX, int *NY)
           {
           
          hipDoubleComplex   *ortkx_d, *ortky_d;  //, *psiV_d;  // declare GPU vector copies cufftDoubleComplex  *psi_d, *ortkx_d;
    
          hipfftDoubleComplex   *psi_d;         

           int NDX = *NX;       
           int NDY = *NY;  
           hipfftHandle plan; 
 
  //  int  BATCH = 1;     // batch Number of transforms of size nx

   //         printf("Kernel CPP FFT 2D\n"); 
       // Allocate memory on GPU
       
          hipMalloc( (void **)&psi_d, sizeof(hipfftDoubleComplex) * NDX*NDY );
    
          hipMalloc( (void **)&ortkx_d, sizeof(hipDoubleComplex) * NDX );
          hipMalloc( (void **)&ortky_d, sizeof(hipDoubleComplex) * NDY );

   //----------  copy vectors from CPU to GPU
      
//      cudaMemcpy( psi_d, psi, sizeof(cufftDoubleComplex) * NX, cudaMemcpyHostToDevice );
 
             /* Initialize CUBLAS */
   
 //          cublasHandle_t handle;
          
   //        cublasCreate(&handle);    // cublasCreate_v2(&handle); 
          
     //    cublasSetVector(NDX*NDY,sizeof(cuDoubleComplex),psi,1, psi_d,1);
       //  cublasSetVector(NDX,sizeof(cuDoubleComplex),ortkx,1, ortkx_d,1);
         //cublasSetVector(NDY,sizeof(cuDoubleComplex),ortky,1, ortky_d,1);

         
         hipMemcpy( psi_d, psi, sizeof(hipfftDoubleComplex)*NDX*NDY, hipMemcpyHostToDevice );

         hipMemcpy( ortkx_d, ortkx, sizeof(hipDoubleComplex) * NDX, hipMemcpyHostToDevice );
         hipMemcpy( ortky_d, ortky, sizeof(hipDoubleComplex) * NDY, hipMemcpyHostToDevice );
   
        /* Create a 2D FFT plan. */
  
             hipfftPlan2d(&plan, NDY, NDX, HIPFFT_Z2Z);

      /* Use the CUFFT plan to transform the signal in place. */
           
                       
            hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_FORWARD);

   // call function on GPU 
               // mltpl_2D<<<blocks,threads>>>(psiV_d, psi_d, ortkx_d, ortky_d,NDX, NDY);

//              printf("Cufft forvard \n"); 
               
                mltpl_2D<<<blocks,threads>>>( psi_d, ortkx_d, ortky_d, NDX, NDY);

//             printf("Kernel 2D\n"); 
 
 //----------------------- kernel on CPU -----------------------------------------
//------------------------PROOF - rabotaet pravil'no -----------------------
/*
             cudaMemcpy(psi, psi_d, sizeof(cuDoubleComplex) * NDX*NDY, cudaMemcpyDeviceToHost);
            for (int jd = 0; jd < NDY; ++jd ) 
            {
             for (int id = 0; id < NDX; ++id ) 
             {
               int ind =id+NDX*jd;
             psi[ind]= cuCmul (cuCadd(ortkx[id], ortky[jd] ),  psi[ind] );
             }
            }
            
            cudaMemcpy( psi_d, psi, sizeof(cuDoubleComplex)*NDX*NDY, cudaMemcpyHostToDevice );

*/
//--------------------------------------------------------------------------------------------

/*           
  Do i=1,NDX
         Do j=1,NDY*NDZ
           ind=i+NDX*(J-1)
            psi(ind)=(ort_kx(i)**2+ ort_ky(j)**2)/(2*Time_mass)/NDX/NDY*
     &            psi(ind)
         EndDo
      EndDo 
 */           
                 
            hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_BACKWARD);

  //           printf("Cufft-Inverse\n"); 
  
      // copy vector back from GPU to CPU  
       
      //      cublasGetVector(NDX*NDY,sizeof(cuDoubleComplex),psi_d,1, psi,1); 
            
             hipMemcpy(psi, psi_d, sizeof(hipfftDoubleComplex) * NDX*NDY, hipMemcpyDeviceToHost);

//                 printf("Copy from device psi[NDX*NDY-1]= %8.6f\n", psi[NDX*NDY-1]); 

                     
   // copy vectors back from GPU to CPU
 //  cudaMemcpy( a, a_d, sizeof(float) * N, cudaMemcpyDeviceToHost );
  // cudaMemcpy( b, b_d, sizeof(float) * N, cudaMemcpyDeviceToHost );

   // free GPU memory
     
           hipfftDestroy(plan);
  
           hipFree(psi_d);
       
           hipFree(ortkx_d);
           hipFree(ortky_d);

        // cublasDestroy(handle);          //  cublasDestroy_v2(handle);
           
        

       return;

         }
