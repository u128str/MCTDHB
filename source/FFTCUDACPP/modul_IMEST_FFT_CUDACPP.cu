#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>


#include <iostream>

 #include <complex>




#define imin(a,b) (a<b?a:b)     

 


__global__ void  prod( hipDoubleComplex *vec2, hipDoubleComplex *a, int NDIM)
 {    
            
          int id = threadIdx.x  + blockIdx.x* blockDim.x;
       hipDoubleComplex  d;
     // make_cuDoubleComplex(1.0/NDIM,0.0); 
         d.x=(1.0/NDIM);
         d.y=0.0;
      
      while(id<NDIM)
      {
                        
            vec2[id]= hipCmul (hipCmul(a[id], d) ,  vec2[id] );

  //        vec2[id]= cuCmul (a[id] / NDIM ,  vec2[id] );

 
         id += blockDim.x*gridDim.x;

      }
  } 


/*--------- function called from main fortran programn ---------------*/

extern "C" void kernel_imestfft_(hipDoubleComplex *wsl,hipDoubleComplex *psi,int *NX,int *NY,int *NZ,  int *Xcase,hipDoubleComplex *vtilde)
{
           hipDoubleComplex  *psi_d, *vtilde_d;

        int NDX = *NX;
        int NDY = *NY;
        int NDZ = *NZ;
        int NDIM= NDX*NDY*NDZ;
        int dcase= *Xcase;

// ---- CUDA  variables: ----------------------------------------------------------------------- 

            const int threads = imin(64,NDIM);   //dim3 (16,1,1);
            const int blocks  = imin(32, NDIM/threads );  // ;  //imin(8, (M+threads)/threads ); 
//----------------------------------------------------------------------------------------------

         hipfftHandle plan;
         hipMalloc( (void **)&psi_d, sizeof(hipDoubleComplex) * NDIM );
         hipMalloc( (void **)&vtilde_d, sizeof(hipDoubleComplex) *NDIM);

         hipMemcpy( psi_d,       psi, sizeof(hipDoubleComplex)*NDIM, hipMemcpyHostToDevice );
         hipMemcpy( vtilde_d, vtilde, sizeof(hipDoubleComplex)*NDIM, hipMemcpyHostToDevice );

//                 printf("Kernel GPU   \n");  

      if (dcase == 1)  hipfftPlan1d(&plan, NDX, HIPFFT_Z2Z, 1); 
     
     if (dcase == 2)   hipfftPlan2d(&plan, NDY, NDX,      HIPFFT_Z2Z); 
   
     if (dcase == 3)   hipfftPlan3d(&plan, NDZ, NDY, NDX, HIPFFT_Z2Z); 

          hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_FORWARD);

          prod<<<blocks,threads>>>( psi_d, vtilde_d, NDIM); 

          hipfftExecZ2Z(plan,  psi_d,  psi_d, HIPFFT_BACKWARD);


        /*  copy vectors from GPU to CPU   */
           
            hipMemcpy(wsl, psi_d, sizeof(hipDoubleComplex) * NDIM, hipMemcpyDeviceToHost);

           hipfftDestroy(plan);
           hipFree(psi_d);
           hipFree(vtilde_d);
       return;

}
